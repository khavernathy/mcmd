#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#if __CUDA_ARCH__ < 200
/* this is an explicit definition for atomicAdd, to be safe */
__device__ double atomicAdd(double* address, double val)
{
 unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;
  do { assumed = old;
  old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed))); // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN) 
  }
  while (assumed != old);
  return __longlong_as_double(old);
}
#endif

// minimal data to send to GPU. this is all that's needed to calc forces.
typedef struct atom_t {
    double pos[3]={0,0,0};
    double eps=0; // lj
    double sig=0; // lj
    double charge=0;
    double f[3]={0,0,0}; // force
    int molid=0;
    int frozen=0;
    double u[3] = {0,0,0}; // dipole
    double polar=0; // polarizability
} cuda_atom;


__global__
void calculateForceKernel(cuda_atom * atom_list, int N, double cutoffD, double * basis, double * reciprocal_basis, int pformD, double ewald_alpha, int kmax, int kspace, double polar_damp) {
    // define thread id
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    // only run for real atoms (no ghost threads)
    if(i<N){   
        const register cuda_atom anchoratom = atom_list[i];
        const int pform = pformD;
        const double damp = polar_damp;
        const double alpha = ewald_alpha;
        const double cutoff=cutoffD;
        register double rimg, rsq;
        const double sqrtPI=sqrt(M_PI);
        double d[3], di[3], img[3], dimg[3],r,r2,ri,ri2;
        int q,j,n;
        double sig,eps,r6,s6,u[3]={0,0,0};
        register double af[3] = {0,0,0}; // accumulated forces for anchoratom
        double holder,chargeprod; // for ES force    
        __syncthreads();
        // order N (generally, IF N_atoms < N_cores) instead of N^2 bc 
        // this runs on all GPU cores at once 

        // if LJ 
        if (pform == 0 || pform == 1 || pform == 2) {
        for (j=i+1;j<N;j++) {

           if (anchoratom.molid == atom_list[j].molid) continue; // skip same molecule 
            if (anchoratom.frozen && atom_list[j].frozen) continue; // skip frozens            
            
            // LB mixing
            sig = anchoratom.sig;
            if (sig != atom_list[j].sig) sig = 0.5*(sig+atom_list[j].sig);
            eps = anchoratom.eps;
            if (eps != atom_list[j].eps) eps = sqrt(eps * atom_list[j].eps);
            if (sig == 0 || eps == 0) continue;

            // get R (nearest image)
            for (n=0;n<3;n++) d[n] = anchoratom.pos[n] - atom_list[j].pos[n];
            for (n=0;n<3;n++) {
                img[n]=0;
                for (q=0;q<3;q++) {
                    img[n] += reciprocal_basis[n*3+q]*d[q];
                }
                img[n] = rint(img[n]);
            }
            for (n=0;n<3;n++) {
                di[n] = 0;
                for (q=0;q<3;q++) {
                    di[n] += basis[n*3+q]*img[q];
                }
            }
            for (n=0;n<3;n++) di[n] = d[n] - di[n];
            r2=0;ri2=0;
            for (n=0;n<3;n++) {
                r2 += d[n]*d[n];
                ri2 += di[n]*di[n];
            }
            r = sqrt(r2);
            ri = sqrt(ri2);
            if (ri != ri) {
                rimg=r;
                rsq=r2;
                for (n=0;n<3;n++) dimg[n] = d[n];
            } else {
                rimg=ri;
                rsq=ri2;
                for (n=0;n<3;n++) dimg[n] = di[n];
            }
            // distance is now rimg
    
            if (rimg <= cutoff) {
                r6 = rsq*rsq*rsq;
                s6 = sig*sig;
                s6 *= s6 * s6;
        
                for (n=0;n<3;n++) {
                    holder = 24.0*dimg[n]*eps*(2*(s6*s6)/(r6*r6*rsq) - s6/(r6*rsq));
                    atomicAdd(&(atom_list[j].f[n]), -holder); 
                    af[n] += holder;      
                }
            }
        } // end pair j
        
        // finally add the accumulated forces (stored on register) to the anchor atom
        for (n=0;n<3;n++)
            atomicAdd(&(atom_list[i].f[n]), af[n]);
        
        } // end if LJ
        // ==============================================================================
        // Now handle electrostatics
        if (pform == 1 || pform == 2) {
            for (n=0;n<3;n++) af[n]=0; // reset register-stored force for anchoratom.
            double invV;
            int l[3], p, q; double k[3], k_sq, fourPI = 4.0*M_PI;
            invV =  basis[0] * (basis[4]*basis[8] - basis[7]*basis[5] );
            invV += basis[3] * (basis[7]*basis[2] - basis[1]*basis[8] );
            invV += basis[6] * (basis[1]*basis[5] - basis[5]*basis[2] );
            invV = 1.0/invV;

           for (j=0;j<N;j++) {
                if (anchoratom.frozen && atom_list[j].frozen) continue; // don't do frozen pairs
                if (anchoratom.charge == 0 || atom_list[j].charge == 0) continue; // skip 0-force
                if (i==j) continue; // don't do atom with itself

               // get R (nearest image)
            for (n=0;n<3;n++) d[n] = anchoratom.pos[n] - atom_list[j].pos[n];
            for (n=0;n<3;n++) {
                img[n]=0;
                for (q=0;q<3;q++) {
                    img[n] += reciprocal_basis[n*3+q]*d[q];
                }
                img[n] = rint(img[n]);
            }
            for (n=0;n<3;n++) {
                di[n] = 0;
                for (q=0;q<3;q++) {
                    di[n] += basis[n*3+q]*img[q];
                }
            }
            for (n=0;n<3;n++) di[n] = d[n] - di[n];
            r2=0;ri2=0;
            for (n=0;n<3;n++) {
                r2 += d[n]*d[n];
                ri2 += di[n]*di[n];
            }
            r = sqrt(r2);
            ri = sqrt(ri2);
            if (ri != ri) {
                rimg=r;
                rsq=r2;
                for (n=0;n<3;n++) dimg[n] = d[n];
            } else {
                rimg=ri;
                rsq=ri2;
                for (n=0;n<3;n++) dimg[n] = di[n];
            }

            // real-space
            if (rimg <= cutoff && (anchoratom.molid < atom_list[j].molid)) { // non-duplicated pairs, not intramolecular, not beyond cutoff
                chargeprod = anchoratom.charge * atom_list[j].charge;
                for (n=0;n<3;n++) u[n] = dimg[n]/rimg;
                for (n=0;n<3;n++) {
                    holder = -((-2.0*chargeprod*alpha*exp(-alpha*alpha*rsq))/(sqrtPI*rimg) - (chargeprod*erfc(alpha*rimg)/rsq))*u[n];
                    af[n] += holder;
                    atomicAdd(&(atom_list[j].f[n]), -holder);                
                }
            }
            // k-space
            if (kspace && (anchoratom.molid < atom_list[j].molid)) {
               chargeprod = anchoratom.charge * atom_list[j].charge;
                
               for (n=0; n<3; n++) {
                    for (l[0] = 0; l[0] <= kmax; l[0]++) {
                    for (l[1] = (!l[0] ? 0 : -kmax); l[1] <= kmax; l[1]++) {
                    for (l[2] = ((!l[0] && !l[1]) ? 1 : -kmax); l[2] <= kmax; l[2]++) {
                        // skip if norm is out of sphere
                        if (l[0]*l[0] + l[1]*l[1] + l[2]*l[2] > kmax*kmax) continue;
                        /* get reciprocal lattice vectors */
                        for (p=0; p<3; p++) {
                            for (q=0, k[p] = 0; q < 3; q++) {
                            k[p] += 2.0*M_PI*reciprocal_basis[3*q+p] * l[q];
                            }
                        }
                        k_sq = k[0]*k[0] + k[1]*k[1] + k[2]*k[2];

                        holder = chargeprod * invV * fourPI * k[n] *
                        exp(-k_sq/(4*alpha*alpha))*
                        sin(k[0]*dimg[0] + k[1]*dimg[1] + k[2]*dimg[2])/k_sq * 2; // times 2 b/c half-Ewald sphere

                        af[n] += holder;
                        atomicAdd(&(atom_list[j].f[n]), -holder);

                    } // end for l[2], n
                    } // end for l[1], m
                    } // end for l[0], l
                } // end 3d 
            }

            } // end pair loop j 

            // finally add ES contribution to anchor-atom
            for (n=0;n<3;n++) atomicAdd(&(atom_list[i].f[n]), af[n]);
        } // end ES component
        // ============================================================
        // Polarization
        if (pform == 2) {
            double common_factor, r, rinv, r2, r2inv, r3, r3inv, r5inv, r7inv;
            double x2,y2,z2,x,y,z;
            double udotu, ujdotr, uidotr;
            const double cc2inv = 1.0/(cutoff*cutoff);
            double t1,t2,t3,p1,p2,p3,p4,p5;
            const double u_i[3] = {anchoratom.u[0], anchoratom.u[1], anchoratom.u[2]};
            double u_j[3];
            // loop all pair atoms
            for (int j=i+1; j<N; j++) {
                for (n=0;n<3;n++) af[n] = 0; // reset local force for this pair.
                if (anchoratom.molid == atom_list[j].molid) continue; // no same-molecule
                // get R (nearest image)
		        for (n=0;n<3;n++) d[n] = anchoratom.pos[n] - atom_list[j].pos[n];
		        for (n=0;n<3;n++) {
		            img[n]=0;
		            for (q=0;q<3;q++) {
		                img[n] += reciprocal_basis[n*3+q]*d[q];
		            }
		            img[n] = rint(img[n]);
		        }
		        for (n=0;n<3;n++) {
		            di[n] = 0;
		            for (q=0;q<3;q++) {
		                di[n] += basis[n*3+q]*img[q];
		            }
		        }
		        for (n=0;n<3;n++) di[n] = d[n] - di[n];
		        r2=0;ri2=0;
		        for (n=0;n<3;n++) {
		            r2 += d[n]*d[n];
		            ri2 += di[n]*di[n];
		        }
		        r = sqrt(r2);
		        ri = sqrt(ri2);
		        if (ri != ri) {
		            rimg=r;
		            rsq=r2;
		            for (n=0;n<3;n++) dimg[n] = d[n];
		        } else {
		            rimg=ri;
		            rsq=ri2;
		            for (n=0;n<3;n++) dimg[n] = di[n];
		        }
                // got pair displacements

                if (rimg > cutoff) continue; // skip outside cutoff
                r = rimg;
                x = dimg[0]; y = dimg[1]; z = dimg[2];
                x2 = x*x;
                y2 = y*y;
                z2 = z*z;
                r2 = r*r;
                r3 = r2*r;
                rinv = 1./r;
                r2inv = rinv*rinv;
                r3inv = r2inv*rinv;
                for (n=0;n<3;n++) u_j[n] = atom_list[j].u[n];

                // (1) u_i -- q_j
                if (atom_list[j].charge != 0 && anchoratom.polar != 0) {
                    common_factor = atom_list[j].charge * r3inv;

		            af[0] += common_factor*((u_i[0]*(r2inv*(-2*x2 + y2 + z2) - cc2inv*(y2 + z2))) + (u_i[1]*(r2inv*(-3*x*y) + cc2inv*x*y)) + (u_i[2]*(r2inv*(-3*x*z) + cc2inv*x*z)));

		            af[1] += common_factor*(u_i[0]*(r2inv*(-3*x*y) + cc2inv*x*y) + u_i[1]*(r2inv*(-2*y2 + x2 + z2) - cc2inv*(x2 + z2)) + u_i[2]*(r2inv*(-3*y*z) + cc2inv*y*z));

		            af[2] += common_factor*(u_i[0]*(r2inv*(-3*x*z) + cc2inv*x*z) + u_i[1]*(r2inv*(-3*y*z) + cc2inv*y*z) + u_i[2]*(r2inv*(-2*z2 + x2 + y2) - cc2inv*(x2 + y2)));
 
                }

                // (2) u_j -- q_i
                if (anchoratom.charge != 0 && atom_list[j].polar != 0) {
                    common_factor = anchoratom.charge * r3inv;

                    af[0] -= common_factor*((u_j[0]*(r2inv*(-2*x2 + y2 + z2) - cc2inv*(y2 + z2))) + (u_j[1]*(r2inv*(-3*x*y) + cc2inv*x*y)) + (u_j[2]*(r2inv*(-3*x*z) + cc2inv*x*z)));

                    af[1] -= common_factor*(u_j[0]*(r2inv*(-3*x*y) + cc2inv*x*y) + u_j[1]*(r2inv*(-2*y2 + x2 + z2) - cc2inv*(x2 + z2)) + u_j[2]*(r2inv*(-3*y*z) + cc2inv*y*z));

                    af[2] -= common_factor*(u_j[0]*(r2inv*(-3*x*z) + cc2inv*x*z) + u_j[1]*(r2inv*(-3*y*z) + cc2inv*y*z) + u_j[2]*(r2inv*(-2*z2 + x2 + y2) - cc2inv*(x2 + y2)));
                }

                // (3) u_i -- u_j
                if (anchoratom.polar != 0 && atom_list[j].polar != 0) {
                    r5inv = r2inv*r3inv;
                    r7inv = r5inv*r2inv;
                    udotu = u_i[0]*u_j[0] + u_i[1]*u_j[1] + u_i[2]*u_j[2];
                    uidotr = u_i[0]*dimg[0] + u_i[1]*dimg[1] + u_i[2]*dimg[2];
                    ujdotr = u_j[0]*dimg[0] + u_j[1]*dimg[1] + u_j[2]*dimg[2];                    

                    t1 = exp(-damp*r);
                    t2 = 1. + damp*r + 0.5*damp*damp*r2;
                    t3 = t2 + damp*damp*damp*r3/6.;
                    p1 = 3*r5inv*udotu*(1. - t1*t2) - r7inv*15.*uidotr*ujdotr*(1. - t1*t3);
                    p2 = 3*r5inv*ujdotr*(1. - t1*t3);
                    p3 = 3*r5inv*uidotr*(1. - t1*t3);
                    p4 = -udotu*r3inv*(-t1*(damp*rinv + damp*damp) + rinv*t1*damp*t2);
                    p5 = 3*r5inv*uidotr*ujdotr*(-t1*(rinv*damp + damp*damp + 0.5*r*damp*damp*damp) + rinv*t1*damp*t3);

                    af[0] += p1*x + p2*u_i[0] + p3*u_j[0] + p4*x + p5*x;
                    af[1] += p1*y + p2*u_i[1] + p3*u_j[1] + p4*y + p5*y;
                    af[2] += p1*z + p2*u_i[2] + p3*u_j[2] + p4*z + p5*z;
                }

                // apply Newton for pair.
                for (n=0;n<3;n++) {
                    atomicAdd(&(atom_list[i].f[n]), af[n]);
                    atomicAdd(&(atom_list[j].f[n]), -af[n]);    
                }

		    } // end pair loop with atoms j
        } // end polarization forces

    } // end if i<n (all threads)
}


__global__
void calculateForceNopbcKernel(cuda_atom * atom_list, int N, int pformD) {
    // define thread id
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    // only run for real atoms (no ghost threads)
    if(i<N){   
        const register cuda_atom anchoratom = atom_list[i];
       const int pform = pformD;
        const double cutoff=10.; // default 10 A for no-pbc cutoff.
        double d[3], r,r2;
        int j,n;
        double sig,eps,r6,s6,u[3]={0,0,0};
        register double af[3] = {0,0,0}; // accumulated forces for anchoratom
        double holder,chargeprod; // for ES force    
        __syncthreads();

        // if LJ 
        if (pform == 0 || pform == 1) {
        for (j=i+1;j<N;j++) {

           if (anchoratom.molid == atom_list[j].molid) continue; // skip same molecule 
            if (anchoratom.frozen && atom_list[j].frozen) continue; // skip frozens            

           
            // get R (nearest image)
            for (n=0;n<3;n++) d[n] = anchoratom.pos[n] - atom_list[j].pos[n];
            r2=0;
            for (n=0;n<3;n++) {
                r2 += d[n]*d[n];
            }
            r = sqrt(r2);
               
                if (r <= cutoff) {
           
                 sig = anchoratom.sig;
                if (sig != atom_list[j].sig) sig = 0.5*(sig+atom_list[j].sig);
                eps = anchoratom.eps;
                if (eps != atom_list[j].eps) eps = sqrt(eps * atom_list[j].eps);

                if (sig == 0 || eps == 0) continue;
                
                r6 = r2*r2*r2;
                s6 = sig*sig;
                s6 *= s6 * s6;
        
                    for (n=0;n<3;n++) {
                        holder = 24.0*d[n]*eps*(2*(s6*s6)/(r6*r6*r2) - s6/(r6*r2));
                        atomicAdd(&(atom_list[j].f[n]), -holder); 
                        af[n] += holder;      
                    }
                }

        } // end pair j
        
        // finally add the accumulated forces (stored on register) to the anchor atom
        for (n=0;n<3;n++)
            atomicAdd(&(atom_list[i].f[n]), af[n]);
        
        } // end if LJ
        // ==============================================================================
        // Now handle electrostatics
        if (pform == 1) {
            for (n=0;n<3;n++) af[n]=0; // reset register-stored force for anchoratom.
           for (j=i+1;j<N;j++) {
                if (anchoratom.frozen && atom_list[j].frozen) continue; // don't do frozen pairs
                if (anchoratom.charge == 0 || atom_list[j].charge == 0) continue; // skip 0-force
                if (anchoratom.molid == atom_list[j].molid) continue; // don't do molecule with itself

               // get R (nearest image)
            for (n=0;n<3;n++) d[n] = anchoratom.pos[n] - atom_list[j].pos[n];
            r2=0;
            for (n=0;n<3;n++) {
                r2 += d[n]*d[n];
            }
            r = sqrt(r2);

            if (r <= cutoff)  { //&& (anchoratom.molid < atom_list[j].molid)) { // non-duplicated pairs, not intramolecular, not beyond cutoff
                chargeprod = anchoratom.charge * atom_list[j].charge;
                for (n=0;n<3;n++) u[n] = d[n]/r;
                for (n=0;n<3;n++) {
                    holder = chargeprod/r2 * u[n];
                    af[n] += holder;
                    atomicAdd(&(atom_list[j].f[n]), -holder);                
                }
            }

            } // end pair loop j 

            // finally add ES contribution to anchor-atom
            for (n=0;n<3;n++) atomicAdd(&(atom_list[i].f[n]), af[n]);
        } // end ES component

    } // end if i<n (all threads)
} // end no-pbc force


void CUDA_force(System &system) {

    const int N = (int)system.constants.total_atoms;
    const int block_size = system.constants.cuda_block_size; 
    const int atoms_array_size=sizeof(cuda_atom)*N;
    int index=0;

    // if polarization force needed, get dipoles on CPU first
    if (system.constants.potential_form == POTENTIAL_LJESPOLAR) {
        if (system.constants.ensemble == ENSEMBLE_UVT) {
            thole_resize_matrices(system); // only if N can change
        }
        thole_amatrix(system); // populate A matrix
        thole_field(system); // calculate electric field
        int num_iterations = thole_iterative(system); // calculate dipoles
            system.stats.polar_iterations.value = (double)num_iterations;
            system.stats.polar_iterations.calcNewStats();
            system.constants.dipole_rrms = get_dipole_rrms(system);
    }

    cuda_atom H[N]; // host atoms
    cuda_atom *D; // device atoms (gpu)
    for (int i=0;i<system.molecules.size();i++) {
        for (int j=0;j<system.molecules[i].atoms.size();j++) {
            H[index].molid = i;
            H[index].sig = system.molecules[i].atoms[j].sig;
            H[index].eps = system.molecules[i].atoms[j].eps;
            H[index].charge = system.molecules[i].atoms[j].C;
            if (system.constants.potential_form == POTENTIAL_LJESPOLAR)
                H[index].polar = system.molecules[i].atoms[j].polar;
            for (int n=0;n<3;n++) {
                H[index].pos[n] = system.molecules[i].atoms[j].pos[n];       
                H[index].f[n] = 0; // initialize to zero
                if (system.constants.potential_form == POTENTIAL_LJESPOLAR) {
                    H[index].u[n] = system.molecules[i].atoms[j].dip[n];
                }
            }
            H[index].frozen = system.molecules[i].atoms[j].frozen;     
            index++;       
        }
    }

    int bs = sizeof(double)*9;
    double *basis;
    double *reciprocal_basis;
    basis = (double*)malloc(bs);
    reciprocal_basis = (double*)malloc(bs);
    double *dbasis;
    double *dreciprocal_basis;

    for (int p=0;p<3;p++) {
        for (int q=0;q<3;q++) {
            basis[3*q+p] = system.pbc.basis[p][q]; 
            reciprocal_basis[3*q+p] = system.pbc.reciprocal_basis[p][q]; 
        }
    }

    // allocate memory on GPU
    hipMalloc((void**) &dbasis, bs);
    hipMemcpy(dbasis, basis, bs, hipMemcpyHostToDevice);
    hipMalloc((void**) &dreciprocal_basis, bs);
    hipMemcpy(dreciprocal_basis, reciprocal_basis, bs, hipMemcpyHostToDevice); 
    hipMalloc((void**) &D, atoms_array_size);
    hipMemcpy(D, H, atoms_array_size, hipMemcpyHostToDevice);

    // grid elements
    int dimGrid = ceil((double)N/block_size);
    int dimBlock = block_size;   

    // assign potential form for force calculator
    int pform,theval=system.constants.potential_form;
    if (theval == POTENTIAL_LJ || theval == POTENTIAL_LJES || theval == POTENTIAL_LJESPOLAR)
        pform=0;
    if (theval == POTENTIAL_LJES || theval == POTENTIAL_LJESPOLAR)
        pform=1;
    if (theval == POTENTIAL_LJESPOLAR)
        pform=2;

    // calculate forces on GPU !!
    calculateForceKernel<<< dimGrid, dimBlock >>>(D,N,system.pbc.cutoff, dbasis, dreciprocal_basis, pform, system.constants.ewald_alpha, system.constants.ewald_kmax, system.constants.kspace_option, system.constants.polar_damp);
    // make sure the threads are synced so we don't overflow
    hipDeviceSynchronize();
    // copy device data back to host
    hipMemcpy(H, D, atoms_array_size, hipMemcpyDeviceToHost);

    index=0;
    for (int i=0;i<system.molecules.size();i++) {
        for (int j=0;j<system.molecules[i].atoms.size();j++) {
            for (int n=0;n<3;n++) {
                system.molecules[i].atoms[j].force[n] = H[index].f[n];
            }     
            index++;       
        }
    }

    // clean up -- so we don't have a memory leak
     hipFree(D);
     hipFree(dbasis);
     hipFree(dreciprocal_basis);
     free(basis);
     free(reciprocal_basis);

}

void CUDA_force_nopbc(System &system) {

/*
    needs works to be fully functional.
*/


    const int N = (int)system.constants.total_atoms;
    const int block_size = system.constants.cuda_block_size; 
    const int atoms_array_size=sizeof(cuda_atom)*N;
    int index=0;

    cuda_atom H[N]; // host atoms
    cuda_atom *D; // device atoms (gpu)
    for (int i=0;i<system.molecules.size();i++) {
        for (int j=0;j<system.molecules[i].atoms.size();j++) {
            H[index].molid = i;
            H[index].sig = system.molecules[i].atoms[j].sig;
            H[index].eps = system.molecules[i].atoms[j].eps;
            H[index].charge = system.molecules[i].atoms[j].C;
            for (int n=0;n<3;n++) {
                H[index].pos[n] = system.molecules[i].atoms[j].pos[n];       
                H[index].f[n] = 0; // initialize to zero
            }
            H[index].frozen = system.molecules[i].atoms[j].frozen;     
            index++;       
        }
    }

    // allocate memory on GPU
    hipMalloc((void**) &D, atoms_array_size);
    hipMemcpy(D, H, atoms_array_size, hipMemcpyHostToDevice);

    // grid elements
    int dimGrid = ceil((double)N/block_size);
    int dimBlock = block_size;   

    // assign potential form for force calculator
    int pform,theval=system.constants.potential_form;
    if (theval == POTENTIAL_LJ || theval == POTENTIAL_LJES || theval == POTENTIAL_LJESPOLAR)
        pform=0;
    if (theval == POTENTIAL_LJES || theval == POTENTIAL_LJESPOLAR)
        pform=1;

    calculateForceNopbcKernel<<< dimGrid, dimBlock >>>(D,N, pform);
    // make sure the threads are synced so we don't overflow
    hipDeviceSynchronize();
    // copy device data back to host
    hipMemcpy(H, D, atoms_array_size, hipMemcpyDeviceToHost);

    //for (int i=0;i<N;i++) printf("H[%i] force0 = %f\n", i, H[i].f[0]);
    index=0;
    for (int i=0;i<system.molecules.size();i++) {
        for (int j=0;j<system.molecules[i].atoms.size();j++) {
            for (int n=0;n<3;n++) {
                system.molecules[i].atoms[j].force[n] = H[index].f[n];
            }     
            index++;       
        }
    }

     hipFree(D);
}
